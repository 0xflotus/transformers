#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define MAX_CAP 4
#define MAX_SEQ 2048

inline __device__ float gelu(const float x)
{
    const float sqrt_param = 0.79788456080286535587989211986876f;
    const float mul_param = 0.044715;
    return x * 0.5f * (1.0f + tanhf(sqrt_param * (x + mul_param * x * x * x)));
}

__global__ void fused_bias_gelu(float* input,
                                const float* bias,
                                int total_count,
                                int intermediate_size)
{
    float4* input_cast = reinterpret_cast<float4*>(input);
    const float4* bias_cast = reinterpret_cast<const float4*>(bias);
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float4 data = input_cast[offset];
        float4 bias_data = bias_cast[offset % intermediate_size];

        data.x += bias_data.x;
        data.y += bias_data.y;
        data.z += bias_data.z;
        data.w += bias_data.w;

        data.x = gelu(data.x);
        data.y = gelu(data.y);
        data.z = gelu(data.z);
        data.w = gelu(data.w);

        input_cast[offset] = data;
    }
}

__global__ void fused_bias_gelu(__half* input,
                                const __half* bias,
                                int total_count,
                                int intermediate_size)
{
#ifdef HALF_PRECISION_AVAILABLE

    float2* input_cast = reinterpret_cast<float2*>(input);
    const float2* bias_cast = reinterpret_cast<const float2*>(bias);

    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    if (offset < total_count) {
        float2 vals_vec = input_cast[offset];
        float2 bias_vec = bias_cast[offset % intermediate_size];

        __half2* vals_half = reinterpret_cast<__half2*>(&vals_vec);
        __half2* bias_half = reinterpret_cast<__half2*>(&bias_vec);

        float2 low_data = __half22float2(vals_half[0]);
        float2 high_data = __half22float2(vals_half[1]);

        float2 low_bias = __half22float2(bias_half[0]);
        float2 high_bias = __half22float2(bias_half[1]);

        low_data.x += low_bias.x;
        low_data.y += low_bias.y;
        high_data.x += high_bias.x;
        high_data.y += high_bias.y;

        low_data.x = gelu(low_data.x);
        low_data.y = gelu(low_data.y);
        high_data.x = gelu(high_data.x);
        high_data.y = gelu(high_data.y);

        vals_half[0] = __float22half2_rn(low_data);
        vals_half[1] = __float22half2_rn(high_data);

        input_cast[offset] = vals_vec;
    }
#endif
}

template <typename T>
void launch_bias_gelu(T* input,
                      const T* bias,
                      int intermediate_size,
                      int batch_size)
{
    int total_count = batch_size * (intermediate_size / 4);
    int threads = 1024;  // intermediate_size / iterations / 4;
    dim3 block_dims(threads);
    dim3 grid_dims(((total_count - 1) / 1024 + 1));  // (batch_size);

    fused_bias_gelu<<<grid_dims, block_dims>>>(
        input, bias, total_count, intermediate_size / 4);
}

template void launch_bias_gelu<float>(float*, const float*, int, int);
template void launch_bias_gelu<__half>(__half*, const __half*, int, int);
